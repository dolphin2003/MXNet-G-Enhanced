#include "hip/hip_runtime.h"
// this implements a simple convolution neural net: conv-maxpool-fullc
#include <vector>
// header file to use mshadow
#include "mshadow/tensor.h"
// helper function to load mnist dataset
#include "util.h"
// this namespace contains all data structures, functions
using namespace mshadow;
// this namespace contains all operator overloads
using namespace mshadow::expr;

// define operations
struct relu{
  MSHADOW_XINLINE static real_t Map(real_t a) {
    using namespace std;
    return max(a, 0.0f);
  }
};
struct relu_grad {
  MSHADOW_XINLINE static real_t Map(real_t a) {
    return a > 0.0f ? 1.0f : 0.0f;
  }
};

/*! \brief interface for nnet, interfacd allows use to use GPU/CPU implementation in a unified way */
class INNet{
 public:
  virtual void Forward(const Tensor<cpu, 4, real_t>& inbatch, Tensor<cpu, 2, real_t> &oubatch) = 0;
  virtual void Backprop(const Tensor<cpu, 2, real_t>& gradout) = 0;
  virtual void Update(void) = 0;
  virtual ~INNet() {}
};

/*!
 * \brief simple two layer conv-net conv-pool-flat-fullc
 *        this implementation is device invariant
 */
template<typename xpu>
class ConvNet : public INNet {
 public:
  // initialize the network
  ConvNet(int batch_size, int insize, int nchannel, int ksize, int kstride, int psize, int num_out)
      :rnd(0), ksize(ksize), kstride(kstride), psize(psize) {
    // setup stream
    Stream<xpu> *stream = NewStream<xpu>();
    ninput.set_stream(stream);
    nhidden.set_stream(stream);
    nhiddenbak.set_stream(stream);
    npool.set_stream(stream);
    npoolbak.set_stream(stream);
    nflat.set_stream(stream);
    nout.set_stream(stream);
    hbias.set_stream(stream); g_hbias.set_stream(stream);
    obias.set_stream(stream);  g_obias.set_stream(stream);
    Ki2h.set_stream(stream);  g_Ki2h.set_stream(stream);
    Wh2o.set_stream(stream);   g_Wh2o.set_stream(stream);
    tmp_col.set_stream(stream);
    tmp_dst.set_stream(stream);
    // setup nodes
    ninput.Resize(Shape4(batch_size, 1, insize, insize));
    nhidden.Resize(Shape4(batch_size, nchannel, (insize - ksize)/kstride+1, (insize -ksize)/kstride+1));
    nhiddenbak.Resize(nhidden.shape_);
    npool.Resize(Shape4(batch_size, nchannel, (nhidden.size(2)+1-psize)/psize, (nhidden.size(3)+1-psize)/psize));
    npoolbak.Resize(npool.shape_);
    nflat.Resize(Shape2(batch_size, npool.size(1)*npool.size(2)*npool.size(3)));
    nout.Resize(Shape2(batch_size, num_out));
    // setup bias
    hbias.Resize(Shape1(nchannel)); g_hbias.Resize(hbias.shape_);
    obias.Resize(Shape1(num_out));  g_obias.Resize(obias.shape_);
    hbias = 0.0f; obias = 0.0f;
    // setup weights
    Ki2h.Resize(Shape2(nchannel, ksize*ksize));  g_Ki2h.Resize(Ki2h.shape_);
    Wh2o.Resize(Shape2(nflat.size(1), num_out));   g_Wh2o.Resize(Wh2o.shape_);
    rnd.SampleGaussian(&Ki2h, 0, 0.01f);
    rnd.SampleGaussian(&Wh2o, 0, 0.01f);

    printf("conv=%d, pool=%d\n", nhidden.size(3), npool.size(3));
 