#include "hip/hip_runtime.h"
/*!
 * Copyright (c) 2016 by Contributors
 * \file spatial_transformer.cu
 * \brief
 * \author Wei Wu
*/

#include "./spatial_transformer-inl.h"
#include <algorithm>
#if MXNET_USE_CUDNN == 1 && CUDNN_MAJOR == 5
#include "./cudnn_spatial_transformer-inl.h"
#endif  // MXNET_USE_CUDNN && CUDNN_MAJOR

namespace mshadow {
template<typename DType>
__global__ void BilinearSamplingForwardKernel(const int i_c, const int i_h,
                                              const int i_w, const DType* data,
                                              const DType* grid, const int o_n,
                                              const int o_c, const int o_h,
                                              const int o_w, DType* out) {
  for (int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
       index < o_n * o_c * o_h * o_w;
       index += blockDim.x * gridDim.x * gridDim.y) {
    // (n, c, h, w) is the element in out
    int w = index % o_w;
    int h = (index / o_w) % o_h;
    int c = (index / o_w / o_h) % o_c;
    int n = index / o_w / o_h / o_c;
    index_t out_index = n * o_c * o_h * o_w + c * o_h * o_w + h * o_w + w;
    index_t grid_index = n * o_h * o_w * 2 + h * o_w + w;
    DType y